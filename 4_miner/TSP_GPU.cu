#include "hip/hip_runtime.h"
/*
Copyright (c) 2011, Texas State University-San Marcos. All rights reserved.

Redistribution and use in source and binary forms, with or without modification,
are permitted for academic, research, experimental, or personal use provided
that the following conditions are met:

   * Redistributions of source code must retain the above copyright notice,
     this list of conditions and the following disclaimer.
   * Redistributions in binary form must reproduce the above copyright notice,
     this list of conditions and the following disclaimer in the documentation
     and/or other materials provided with the distribution.
   * Neither the name of Texas State University-San Marcos nor the names of its
     contributors may be used to endorse or promote products derived from this
     software without specific prior written permission.

For all other uses, please contact the Office for Commercialization and Industry
Relations at Texas State University-San Marcos <http://www.txstate.edu/ocir/>.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON
ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

Authors: Martin Burtscher and Molly A. O'Neil
*/

#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

#define MULT 1103515245
#define ADD 12345
#define MASK 0x7fffffff

#define CIPOW2 128  /* power of 2 and as large as the largest possible input */

#define TOURS110 (65536*2)

__device__ volatile unsigned int gblkcnt, gcurr;
__device__ volatile unsigned long long best;

/******************************************************************************/
/* Kernels to initialize best tour tracking and GPU state.                    */
/******************************************************************************/

__global__ void ResetKernel()
{
  register int k;

  k = threadIdx.x + blockIdx.x * blockDim.x;
  if (k == 0) {
    best = 0xffffffffffffffffULL;
  }
}

__global__ void Reset110Kernel(int blocksTimesThreads)
{
  gblkcnt = 0;
  gcurr = blocksTimesThreads;
}

/******************************************************************************/
/* The GPU TSP kernel: for each thread, initializes a first climber and       */
/* randomizes a starting tour, then evaluates IHC steps until a local minimum */
/* is found, at which point a new climber is obtained from the worklist.      */
/******************************************************************************/

__global__
__launch_bounds__(1024, 1)
void TSP110Kernel(int *gdist, int *gresult, int touroffset, int cities, int tours, int *gtours)
{
  register int i, j, change, mini, minj, minchange, randx, citiesm1, citiespad;
  register int ti, tj, tiplus1, tjplus1, dist_i_iplus1, ti_p1, mytour, from, to, cost;
  register int *sdist_ti, *sdist_tiplus1, *ltour;
  register unsigned long long mycost, current;
  unsigned char tour[110];  // local memory for coalesced accesses, shifted by one entry relative to gresult
  __shared__ int sdist[110 * 110];

  for (i = threadIdx.x; i < cities * cities; i += blockDim.x) {
    sdist[i] = gdist[i];
  }
  __syncthreads();

  citiesm1 = cities - 1;
  citiespad = (citiesm1 + 33) & (~31);
  mytour = threadIdx.x + blockIdx.x * blockDim.x;
  if (mytour < tours) {
    // Default starting tour for this thread's first climber
    for (i = 0; i < citiesm1; i++) {
      tour[i] = i + 1;
    }
    tour[citiesm1] = 0;

    // Randomize the initial tour
    randx = mytour + touroffset;  // use mytour as random seed
    for (i = 0; i < citiesm1; i++) {
      randx = (MULT * randx + ADD) & MASK;
      j = randx % citiesm1;
      to = tour[i];
      tour[i] = tour[j];
      tour[j] = to;
    }

    do {
      minchange = 0;
      ti = 0;  // tour[-1]
      ti_p1= tour[0];
      // Evaluate the 2-opt moves: Remove edges (i, i+1) and (j, j+1) and replace with (i, j) and (i+1, j+1)
      // Evaluate each city i (except first/last) against all cities from i+2 to last city, thus not evaluating
      // duplicate reversals or adjacent city pairs, which cannot improve tour.
      for (i = 2; i < cities; i++) {  // loop bias = 2, loop from i=1 to i=cities-2
        tiplus1 = ti_p1 * cities;
        dist_i_iplus1 = sdist[ti + ti_p1];
        tj = ti_p1 = tour[i-1];
        sdist_ti = &sdist[ti]; // Save pointers to i and i+1 rows of distance matrix 
        sdist_tiplus1 = &sdist[tiplus1];
#pragma unroll 8
        for (j = i; j < cities; j++) {
          tjplus1 = tour[j];
          // Instead of recomputing new tour length after 2-opt move, just calculate the change from
          // adding edges (i, j) & (i+1, j+1) and removing edges (i, i+1) & (j, j+1)
          change = sdist_ti[tj] + sdist_tiplus1[tjplus1] - dist_i_iplus1 - sdist[tj * cities + tjplus1];
          tj = tjplus1;
          // If tour length reduction is new minimum, save the (i, j) coordinates of the 2-opt move
          if ((minchange > change) && (j < cities)) {
            minchange = change;
            mini = i;
            minj = j;
          }
        }
        ti = tiplus1;
      }

      // If this climber found an improved tour, perform the city ordering swap
      // and continue evaluating with a new IHC step
      if (minchange < 0) {
        // new tour is 0 to mini, minj downto mini+1, minj+1 to cities
        i = mini - 2;
        j = minj - 1;
        while (i < j) {
          to = tour[j];
          tour[j] = tour[i];
          tour[i] = to;
          i++;
          j--;
        }
      } 
      // Otherwise, this climber found a local minimum, so compute the tour cost,
      // record if best solution so far, and get a new climber from the worklist
      else {
        cost = 0;
        from = 0;
        for (i = 0; i < citiesm1; i++) {
          to = tour[i];
          cost += sdist[from * cities + to];
          from = to;
        }
        mycost = cost + sdist[from];
        mycost = (mycost << 32) + mytour + touroffset;

        current = best;
        // Is our local minimum the best solution found?  If so, compare-and-swap to
        // save it as the current best
        while (mycost < current) {
          atomicCAS((unsigned long long *)&best, current, mycost);
          current = best;
        }
        if (mycost == current) {
          ltour = &gtours[mytour * citiespad];
          for (i = 0; i < citiesm1; i++) {
            ltour[i] = tour[i];
          }
        }
        // Get the next climber and randomize a new tour
        mytour = atomicAdd((int *)&gcurr, 1);
        if (mytour < tours) {
          for (i = 0; i < citiesm1; i++) {
            tour[i] = i + 1;
          }
          tour[citiesm1] = 0;

          randx = mytour + touroffset;
          for (i = 0; i < citiesm1; i++) {
            randx = (MULT * randx + ADD) & MASK;
            j = randx % (citiesm1);
            to = tour[i];
            tour[i] = tour[j];
            tour[j] = to;
          }
        }
      }
    } while (mytour < tours);
  }

  __syncthreads();
  if (threadIdx.x == 0) {
    to = gridDim.x - 1;
    if (to == atomicInc((unsigned int *)&gblkcnt, to)) {
      mytour = best & 0xffffffff;
      gresult[0] = best >> 32;
      gresult[1] = 0;
      gresult[2] = mytour;
      mytour %= TOURS110;
      ltour = &gtours[mytour * citiespad];
      for (i = 0; i < citiesm1; i++) {
        gresult[i+3] = ltour[i];
      }
    }
  }
}

/******************************************************************************/
/* Function to read the TSP database input file and initialize the distance   */
/* matrix.                                                                    */
/******************************************************************************/

static int readFile(char *filename, int *dist)
{
  register int i, j, ch, cnt, cities;
  int i1;
  float i2, i3;
  register float *posx, *posy;
  register double dx, dy;
  register FILE *f;
  char str[256];

  f = fopen(filename, "r+t");
  if (f == NULL) {fprintf(stderr, "could not open file %s\n", filename); exit(-1);}

  ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);
  ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);
  ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);

  ch = getc(f);  while ((ch != EOF) && (ch != ':')) ch = getc(f);
  fscanf(f, "%s\n", str);
  cities = atoi(str);
  if (cities == 0) {
    fprintf(stderr, "%d cities\n", cities);
    exit(-1);
  }
  if (cities >= CIPOW2) {
    fprintf(stderr, "%d cities is too large\n", cities);
    exit(-1);
  }
  posx = (float *)malloc(sizeof(float) * cities);
  posy = (float *)malloc(sizeof(float) * cities);
  if ((posx == NULL) || (posy == NULL)) {
    fprintf(stderr, "out of memory\n");
    exit(-1);
  }

  ch = getc(f);  while ((ch != EOF) && (ch != '\n')) ch = getc(f);
  fscanf(f, "%s\n", str);
  if (strcmp(str, "NODE_COORD_SECTION") != 0) {
    fprintf(stderr, "wrong file format\n");
    exit(-1);
  }

  cnt = 0;
  while (fscanf(f, "%d %f %f\n", &i1, &i2, &i3)) {
    posx[cnt] = i2;
    posy[cnt] = i3;
    cnt++;
    if (cnt > cities) {fprintf(stderr, "input too long\n"); exit(-1);}
    if (cnt != i1) {fprintf(stderr, "input line mismatch: expected %d instead of %d\n", cnt, i1); exit(-1);}
  }
  if (cnt != cities) {
    fprintf(stderr, "read %d instead of %d cities\n", cnt, cities);
    exit(-1);
  }
  fscanf(f, "%s", str);
  if (strcmp(str, "EOF") != 0) {
    fprintf(stderr, "didn't see 'EOF' at end of file\n");
    exit(-1);
  }
  fclose(f);

  for (i = 0; i < cities; i++) {
    for (j = 0; j < cities; j++) {
      dx = posx[i] - posx[j];
      dy = posy[i] - posy[j];
      dist[j * cities + i] = dist[i * cities + j] = (int)(sqrt(dx * dx + dy * dy) + 0.5);
    }
    dist[i * cities + i] = 0x3fffffff;  // half of maxint
  }

  free(posx);
  free(posy);

  return cities;
}

/******************************************************************************/
/* Functions to synchronize GPU threads and check for error status, as well   */
/* as to ascertain number of SMs in device and proper architecture version.      */
/******************************************************************************/

static void CudaTest(char *msg)
{
  hipError_t e;

  hipDeviceSynchronize();
  if (hipSuccess != (e = hipGetLastError())) {
    fprintf(stderr, "%s: %d\n", msg, e);
    fprintf(stderr, "%s\n", hipGetErrorString(e));
    exit(-1);
  }
}

static int VerifySystemParameters(int *SMs)
{
  int deviceCount, currentDevice = 0, bestSMArch = 0;
  int maxComputePerf = 0, maxPerfDevice = 0, SMPerMP;
  int archCoresSM[3] = { 1, 8, 32 };
  hipDeviceProp_t deviceProp;
  
  hipGetDeviceCount(&deviceCount);
  if(deviceCount <= 0) {
    fprintf(stderr, "There is no device supporting CUDA\n");
    exit(-1);
  }

  // Find the best SM architecture device
  while(currentDevice < deviceCount) {
    hipGetDeviceProperties(&deviceProp, currentDevice);
    if(deviceProp.major > 0 && deviceProp.major < 9999) {
      bestSMArch = max(bestSMArch, deviceProp.major);
    }
    currentDevice++;
  }

  // Find the best GPU device
  currentDevice = 0;
  while(currentDevice < deviceCount) {
    hipGetDeviceProperties(&deviceProp, currentDevice);
    if(deviceProp.major == 9999 && deviceProp.minor == 9999) {
      SMPerMP = 1;
    } else if (deviceProp.major <= 2) {
      SMPerMP = archCoresSM[deviceProp.major];
    } else { // SM major > 2
      SMPerMP = archCoresSM[2];
    }
    int computePerf = deviceProp.multiProcessorCount * SMPerMP * deviceProp.clockRate;
    if((deviceProp.major == bestSMArch) && (computePerf > maxComputePerf)) {
      maxComputePerf = computePerf;
      maxPerfDevice = currentDevice;
    }
    currentDevice++;
  }

  hipGetDeviceProperties(&deviceProp, maxPerfDevice);
  if(deviceProp.major < 2) {
    fprintf(stderr, "No device found with compute capability 2.0 or above\n");
    exit(-1);
  }

  *SMs = deviceProp.multiProcessorCount;
  return maxPerfDevice;
}

/******************************************************************************/
/* Run function reads input database and launches the GPU kernels.            */
/* Prints to std out: GPU best tour and tour cost, as well as error from      */
/* optimal tour read from input file.                                         */
/******************************************************************************/

void run(char *filename, int tours, int SMs)
{
  int *lgdist, tour, blocks, best;
  int *lgresult, *lgtours, *lscratch;
  int dist[CIPOW2 * CIPOW2];
  int result[3 + CIPOW2];
  int cities;

  cities = readFile(filename, dist);
  printf("%s: %d tours with %d cities each\n", filename, tours, cities);
  
  if (tours < 1) {
     fprintf(stderr, "tour count must be positive\n");
     exit(-1);
  }

  if (hipSuccess != hipMalloc((void **)&lgtours, ((cities + 32) & (~31)) * min(TOURS110, tours) * sizeof(int))) fprintf(stderr, "could not allocate gtours\n");  CudaTest("couldn't allocate gtours");
  if (hipSuccess != hipMalloc((void **)&lscratch, ((cities + 32) & (~31)) * min(TOURS110, tours) * sizeof(int))) fprintf(stderr, "could not allocate scratch\n");  CudaTest("couldn't allocate scratch");
  if (hipSuccess != hipMalloc((void **)&lgresult, sizeof(int) * (cities + 3))) fprintf(stderr, "could not allocate gresult\n");  CudaTest("couldn't allocate gresult");
  if (hipSuccess != hipMalloc((void **)&lgdist, sizeof(int) * cities * cities)) fprintf(stderr, "could not allocate gdist\n");  CudaTest("couldn't allocate gdist");
  if (hipSuccess != hipMemcpy(lgdist, dist, sizeof(int) * cities * cities, hipMemcpyHostToDevice)) fprintf(stderr, "copying of dist to device failed\n");  CudaTest("dist copy to device failed");

  ResetKernel<<<SMs*3, 512>>>();
  best = 0x7fffffff;
  tour = 0;
  if (cities <= 110) {
    blocks = min(tours, TOURS110);
    while (tours > tour) {
      Reset110Kernel<<<1, 1>>>(SMs*2*512);
      TSP110Kernel<<<SMs, 1024>>>(lgdist, lgresult, tour, cities, blocks, lgtours);

      if (hipSuccess != hipMemcpy(result, lgresult, sizeof(int) * 2, hipMemcpyDeviceToHost)) fprintf(stderr, "copying of result from device failed\n");  CudaTest("result copy from device failed");
      if (best > result[0]) {
        best = result[0];
        if (hipSuccess != hipMemcpy(result, lgresult, sizeof(int) * (cities + 3), hipMemcpyDeviceToHost)) fprintf(stderr, "copying of result from device failed\n");  CudaTest("result copy from device failed");
      }

      tour += blocks;
      blocks = min(tours-tour, TOURS110);
    }
  }
  else {
    fprintf(stderr, "city count must be <= 110\n");
    exit(-1);
  }

  printf("GPU min cost = %d\n", best);
  printf("GPU min tour = %d\n", result[2]);

  hipFree(lgtours);
  hipFree(lscratch);
  hipFree(lgresult);
  hipFree(lgdist);
}


/******************************************************************************/
/* MAIN                                                                       */
/* Usage:  ./TSP_GPU <path to input database> <number of climbers>            */
/******************************************************************************/

int main(int argc, char *argv[])
{
  register int climbers, SMs, deviceID;

  if(argc != 3) {
    fprintf(stderr, "usage: %s <path to input database> <number of climbers>\n", argv[0]);
    exit(-1);
  }

  printf("\nTSP_GPU v1.0  Copyright (c) 2011 Texas State University-San Marcos\n");
  
  deviceID = VerifySystemParameters(&SMs);
  hipSetDevice(deviceID);
  CudaTest("initialization");

  hipFuncSetCacheConfig(reinterpret_cast<const void*>(TSP110Kernel), hipFuncCachePreferShared);

  climbers = atoi(argv[2]);
  run(argv[1], climbers, SMs);

  return 0;
}

